#include "hip/hip_runtime.h"

#include <stdio.h>
#include "check.h"
#define N 10

__global__
void add(int *a, int *b, int *c)
{
  int tid = blockIdx.x;
  if (tid < N) {
      c[tid] = a[tid] + b[tid];
  }
}

int main(int argc, char **argv)
{
  int A[N], B[N], C[N];

  int *dev_A, *dev_B, *dev_C;

  // Allocate arrays on the GPU
  CHECK( hipMalloc( (void**)&dev_A, N*sizeof(int)) );
  CHECK( hipMalloc( (void**)&dev_B, N*sizeof(int)) );
  CHECK( hipMalloc( (void**)&dev_C, N*sizeof(int)) );

  // Initialise the arrays on the CPU
  for (int i = 0; i < N; i++) {
      A[i] = -i;
      B[i] = i*i;
  }

  // Copy the input arrays from host to device
  CHECK( hipMemcpy( dev_A, A, N*sizeof(int), hipMemcpyHostToDevice) );
  CHECK( hipMemcpy( dev_B, B, N*sizeof(int), hipMemcpyHostToDevice) );

  // Perform addition via the kernel

  add<<<N,1>>>( dev_A, dev_B, dev_C );

  // Copy the results back from device to host array
  CHECK( hipMemcpy( C, dev_C, N*sizeof(int), hipMemcpyDeviceToHost) );

  // Output result
  for (int i = 0; i < N; i++)
    printf( "%d + %d = %d\n", A[i], B[i], C[i]);

  // Clean up device.
  hipFree( dev_A );
  hipFree( dev_B );
  hipFree( dev_C );

  return 0;
}
