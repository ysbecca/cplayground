#include "hip/hip_runtime.h"

#include <stdio.h>
#include "check.h"
#define N 10

__global__
void reverse(int *A, int *R)
{
  int tid = blockIdx.x;
  if (tid < N) {
      R[N - tid - 1] = A[tid];
  }
}

int main(int argc, char **argv)
{
  int A[N], R[N];

  int *dev_A, *dev_R;

  // Allocate arrays on the GPU
  CHECK( hipMalloc( (void**)&dev_A, N*sizeof(int)) );
  CHECK( hipMalloc( (void**)&dev_R, N*sizeof(int)) );

  // Initialise the arrays on the CPU
  for (int i = 0; i < N; i++) {
      A[i] = i;
  }

  // Copy the input arrays from host to device
  CHECK( hipMemcpy( dev_A, A, N*sizeof(int), hipMemcpyHostToDevice) );
  CHECK( hipMemcpy( dev_R, R, N*sizeof(int), hipMemcpyHostToDevice) );

  // Perform addition via the kernel
  reverse<<<N,1>>>( dev_A, dev_R );

  // Copy the results back from device to host array
  CHECK( hipMemcpy( R, dev_R, N*sizeof(int), hipMemcpyDeviceToHost) );

  // Output result
  for (int i = 0; i < N; i++)
    printf( "%d, %d\n", A[i], R[i]);

  // Clean up device.
  hipFree( dev_A );
  hipFree( dev_R );

  return 0;
}
